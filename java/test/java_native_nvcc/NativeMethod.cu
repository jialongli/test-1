#include "hip/hip_runtime.h"
#include"NativeMethod.h"
#include<stdio.h>
#include<malloc.h>
#include<math.h>
#include<stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>


void check_gpu_error (const char *msg) 
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) { 
	printf("Cuda error: %s: %s\n", msg, hipGetErrorString (err)); 
	exit(0);   
    }
}
__global__ void fun(float *val,int nx,int nz)
{		
	 int id=threadIdx.x+blockDim.x*blockIdx.x;
    
        if(id<nx*nz)
                val[id] = id+1;

}

JNIEXPORT void JNICALL Java_NativeMethod_writefloat
  (JNIEnv *env, jobject obj, jint nx, jint nz){

	float *v,*vv;

      char FN1[250]={"v.dat"};
	FILE *fp;
      fp=fopen(FN1,"wb");

        printf("Rong Tao shinidaye\n");

    	 v=(float*)malloc(nx*nz*sizeof(float));

        hipSetDevice(0);// initialize device, default device=0;
	 check_gpu_error("Failed to initialize device!");

        hipMalloc(&vv, nx*nz*sizeof(float));
        hipMemset(vv, 0, nx*nz*sizeof(float));

        fun<<<(nx*nz+511)/512,512>>>(vv, nx, nz);

       hipMemcpy(v, vv, nx*nz*sizeof(float), hipMemcpyDeviceToHost);

        fwrite(v,sizeof(float),nx*nz, fp);

        fclose(fp);
}
